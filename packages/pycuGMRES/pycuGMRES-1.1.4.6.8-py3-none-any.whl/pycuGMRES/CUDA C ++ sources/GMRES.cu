#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <string>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "GMRES_kernels.cuh"
#include "saveGPU.cuh"
#include "Fast_matvec.cuh"
#include "pycuFunctions.cuh"

void cuSolve_LES(	hipComplex *dev_A,
			hipComplex *dev_B,
			const int m,
			hipsolverHandle_t cusolverH,
			hipComplex *dev_work,
			int *dev_Ipiv,
			int *dev_info,
			timespec *h_computation_times,
			unsigned int *clock_i_p);


void show_norm(	const char *description,
		hipblasHandle_t handle,
		const hipComplex *dev_array,
		const unsigned int size_array);

void show_norm_F(	const char *description,
			hipblasHandle_t handle,
			const float *dev_array,
			const unsigned int size_array);

void pycuGMRES(	
                    bool *dev_mask,
                    hipComplex *dev_solution,
		    const bool for_gradient,
		    const unsigned int h_index_of_max,
		    unsigned int maxiter,
		    const float tolerance,
		    unsigned int *GMRES_n,
		    float *dev_actual_residual,
		    bool *h_res_vs_tol_p,
		    const unsigned int N,
                    hipComplex *dev_gamma_array,
                    const hipfftHandle plan,
                    hipblasHandle_t *handle_p,
                    hipsolverHandle_t *cusolverH_p,
                    devSubsidiary *dev_subs,
                    timespec *h_computation_times
               )
{
	unsigned int clock_i = 0;

	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_0_

	dim3 blocks(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 blocks_M(THREADS_PER_BLOCK_M, THREADS_PER_BLOCK_M);
	dim3 threads(Q, Q);
	dim3 blocksize(512);
	dim3 gridsize(N * N / blocksize.x);

	float h_residual_initial = 0.f;
	float h_actual_residual = 0.f;
	hipComplex h_Jtotal_0j = {};

	hipComplex *dev_orthogonal_basis		 = dev_subs->dev_orthogonal_basis;
	hipComplex *dev_HH			 = NULL;
	hipComplex *dev_Htemp			 = NULL;
	hipComplex *dev_extended			 = NULL;
	hipComplex *dev_vec_resudual		 = NULL;
	hipComplex *dev_H_			 = NULL;
	hipComplex *dev_alpha			 = NULL;
	hipComplex *dev_beta			 = NULL;
	hipComplex *dev_Jtotal			 = NULL;
	hipComplex *dev_cc 			 = NULL;
	hipComplex *dev_Givens_rotation 		 = NULL;
	hipComplex *dev_Givens_rotation_0	 = NULL;
	hipComplex *dev_Givens_rotation_1	 = NULL;
	hipComplex *dev_Givens_rotation_2	 = NULL;
	hipComplex *dev_Givens_rotation_3	 = NULL;
	hipComplex *dev_buffer_LES_cc		 = NULL;

	int *dev_info 				 = dev_subs->dev_info;
	int *dev_Ipiv 				 = NULL;

	unsigned int GMRES_i = 0;
	const unsigned int maxiter_plus_1 = maxiter + 1;

	dev_vec_resudual = dev_H_ = dev_orthogonal_basis + maxiter_plus_1 * N * N;
	dev_Jtotal = dev_vec_resudual + maxiter * maxiter_plus_1;

	dev_extended = dev_HH = dev_Htemp = dev_vec_resudual + N * N;

	dev_cc = dev_extended + maxiter * maxiter;
	dev_Givens_rotation = dev_Jtotal + maxiter_plus_1 * maxiter_plus_1;
	dev_Givens_rotation_3 = dev_Givens_rotation + maxiter_plus_1 * maxiter_plus_1 - 1;
	dev_Givens_rotation_2 = dev_Givens_rotation_3 - 1;
	dev_Givens_rotation_1 = dev_Givens_rotation_2 - maxiter;
	dev_Givens_rotation_0 = dev_Givens_rotation_1 - 1;
	dev_alpha  = dev_Givens_rotation + maxiter_plus_1 * maxiter_plus_1;
	dev_beta   = dev_alpha + 1;
	dev_buffer_LES_cc = dev_beta + 1;

	dev_Ipiv = dev_info + 1;
							
	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_1_ //Initialization
//========================================= BEGIN: get_residual_vector =======================================================
	if (for_gradient)
	{
		G_x_fft_matvec(	(hipComplex *)dev_gamma_array,
				(hipComplex *)dev_solution,
				(hipComplex *)dev_extended,
				(hipfftHandle)plan, N);

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_2_

		_2D_to_1D_compared_kernel <<< blocks, threads >>> (	(bool *)dev_mask,
									(hipComplex *)dev_solution,
									(hipComplex*)dev_extended,
									(hipComplex*)dev_vec_resudual,
									h_index_of_max, N);
	}
	else
	{
		G_x_fft_matvec(	(hipComplex *)dev_gamma_array,
				(bool *)dev_mask,
				(hipComplex *)dev_solution,
				(hipComplex *)dev_extended,
				(hipfftHandle)plan, N);

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_2_

		_2D_to_1D_compared_kernel <<< blocks, threads >>> (	(hipComplex *)dev_solution,
									(hipComplex*)dev_extended,
									(hipComplex*)dev_vec_resudual, N);
	}
	cudacheckSYN();

	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_3_
//========================================== END: get_residual_vector =========================================================
	cublascall(hipblasScnrm2(        (hipblasHandle_t) *handle_p,
					N * N,
		                        (const hipComplex *)dev_vec_resudual, 1,
					(float  *)dev_actual_residual));
	cudacheckSYN();

	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_4_
//============================================= Begin: Condition to iterate ==========================================================
	cudacall(hipMemcpyAsync(&h_residual_initial, dev_actual_residual, sizeof(float), hipMemcpyDeviceToHost));

	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_5_
//=============================================== End: Condition to iterate ===========================================================
//============================================BEGIN:residual_normalization_kernel=======================================================
	residual_normalization_kernel <<< gridsize, blocksize >>> (	(hipComplex *)dev_vec_resudual,
									(float *)dev_actual_residual,
									(hipComplex *)dev_orthogonal_basis);
	cudacheckSYN();

	set_alpha_beta_kernel <<< 4, 1 >>> ((hipComplex *)dev_alpha, (hipComplex *)dev_beta);
	//don't synchronize

	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_6_
//============================================= END:residual_normalization_kernel ==================================================
	if (h_residual_initial > tolerance)
	{
        	fprintf(stderr, "GMRES_i = %u\n", GMRES_i);

		cudacall(hipMemsetAsync((hipComplex *)dev_H_, 0, maxiter_plus_1 * maxiter * sizeof(hipComplex)));		//don't synchronize

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_7_

		if (for_gradient)
		{
			G_x_fft_matvec(	(hipComplex *)dev_gamma_array,
					(hipComplex *)dev_orthogonal_basis,
					(hipComplex *)dev_extended,
					(hipfftHandle)plan, N);

			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_8_

			_2D_to_1D_kernel <<< blocks, threads >>> (	(bool *)dev_mask,
									(hipComplex*)dev_orthogonal_basis,
									(hipComplex *)dev_extended,
									(hipComplex *)dev_orthogonal_basis + N * N, N);
		}
		else
		{
			G_x_fft_matvec(	(hipComplex *)dev_gamma_array,
					(bool *)dev_mask,
					(hipComplex *)dev_orthogonal_basis,
					(hipComplex *)dev_extended,
					(hipfftHandle)plan, N);

			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_8_

			_2D_to_1D_kernel <<< blocks, threads >>> (	(hipComplex*)dev_orthogonal_basis,
									(hipComplex *)dev_extended,
									(hipComplex *)dev_orthogonal_basis + N * N, N);
		}
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_9_

		cublascall(hipblasCdotc(		(hipblasHandle_t) *handle_p, N * N,
						(const hipComplex *)dev_orthogonal_basis, 1,
						(const hipComplex *)dev_orthogonal_basis + N * N, 1,
						(hipComplex *)dev_H_));
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_10_

		weight_subtract_kernel <<< gridsize, blocksize >>> (	(hipComplex *)dev_orthogonal_basis + N * N,
									(hipComplex *)dev_H_,
									(hipComplex *)dev_orthogonal_basis);
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_11_

		cublascall(hipblasScnrm2(        (hipblasHandle_t) *handle_p,
						N * N,
				                (const hipComplex *)dev_orthogonal_basis + N * N, 1,
						(float  *)dev_actual_residual + maxiter));
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_12_
	//============================================== BEGIN: Fill Orthogonal Basis matrix ============================================
		get_complex_divided <<< 3, 1 >>> (	(const float *)dev_actual_residual + maxiter,
							(hipComplex *)dev_H_ + maxiter,
							(float *)dev_actual_residual + maxiter);
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_13_

		cublascall(hipblasCsscal(	(hipblasHandle_t) *handle_p, N * N,
				            	(const float           *)dev_actual_residual + maxiter,
				            	(hipComplex       *)dev_orthogonal_basis + N * N, 1));
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_14_
	//============================================== END: Orthogonal Basis matrix  ==================================================
	//============================================= BEGIN: Create Givens_Rotation_Matrix ========================================
		set_Identity_matrix_kernel <<< dim3(maxiter_plus_1, maxiter_plus_1), dim3(1, 1) >>> ((hipComplex *)dev_Givens_rotation);
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_15_
	//=============================================== END: Create Givens_Rotation_Matrix,  ========================================
	//============================================= BEGIN: Create Jtotal_Matrix ========================================
		set_first_Jtotal_kernel <<< maxiter_plus_1 * maxiter_plus_1 * 2, 1 >>> ((hipComplex *)dev_Jtotal,
											(hipComplex *)dev_H_,
											maxiter,
											maxiter_plus_1);
		cudacheckSYN();

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++);//_16_
	//=============================================== END: Create Jtotal_Matrix,  ========================================
	//===================================================== BEGIN: Update residual ======================================================
		cudacall(hipMemcpy(&h_Jtotal_0j, dev_Jtotal + maxiter_plus_1, sizeof(hipComplex), hipMemcpyDeviceToHost));
		h_actual_residual = h_residual_initial * sqrt( (pow((float)h_Jtotal_0j.x, 2.0f) + pow((float)h_Jtotal_0j.y, 2.f)));
		cudacall(hipMemcpyAsync(dev_actual_residual + 1, &h_actual_residual, sizeof(float), hipMemcpyHostToDevice));

		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_17_
	//======================================================= END: Update residual ======================================================
		GMRES_i ++;

		for(GMRES_i = 1; ((GMRES_i < maxiter)); GMRES_i ++) //(h_actual_residual > tolerance) &&
		{

                	fprintf(stderr, "GMRES_i = %u\n", GMRES_i);

			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_18_

			if (for_gradient)
			{	
				G_x_fft_matvec(	(hipComplex *)dev_gamma_array,
						(hipComplex *)dev_orthogonal_basis + GMRES_i * N * N,
						(hipComplex *)dev_extended,
						(hipfftHandle)plan, N);

				clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_19_

				_2D_to_1D_kernel <<< blocks, threads >>> (	(bool *)dev_mask,
										(hipComplex*)dev_orthogonal_basis + GMRES_i * N * N,
										(hipComplex *)dev_extended,
										(hipComplex *)dev_orthogonal_basis + (GMRES_i + 1) * N * N, N);
			}
			else
			{
				G_x_fft_matvec(	(hipComplex *)dev_gamma_array,
						(bool *)dev_mask,
						(hipComplex *)dev_orthogonal_basis + GMRES_i * N * N,
						(hipComplex *)dev_extended,
						(hipfftHandle)plan, N);

				clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_19_

				_2D_to_1D_kernel <<< blocks, threads >>> (	(hipComplex*)dev_orthogonal_basis + GMRES_i * N * N,
										(hipComplex *)dev_extended,
										(hipComplex *)dev_orthogonal_basis + (GMRES_i + 1) * N * N, N);
			}
			cudacheckSYN();

			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_20_
	
			for(unsigned int j = 0; j < GMRES_i + 1; j++)
			{
				cublascall(hipblasCdotc(	(hipblasHandle_t) *handle_p, N * N,
							(const hipComplex *)dev_orthogonal_basis + j * N * N, 1,
							(const hipComplex *)dev_orthogonal_basis + (GMRES_i + 1) * N * N, 1,
							(hipComplex *)dev_H_ + j * maxiter + GMRES_i));
				cudacheckSYN();
				clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_21_

				weight_subtract_kernel <<< gridsize, blocksize >>> (	(hipComplex *)dev_orthogonal_basis + (GMRES_i + 1) * N * N,
											(hipComplex *)dev_H_ + j * maxiter + GMRES_i,
											(hipComplex *)dev_orthogonal_basis + j * N * N);
				cudacheckSYN();
				clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_22_
			}

			cublascall(hipblasScnrm2((hipblasHandle_t) *handle_p,
						N * N,
						(const hipComplex *)dev_orthogonal_basis + (GMRES_i + 1) * N * N, 1,
						(float  *)dev_actual_residual + maxiter));
			cudacheckSYN();

			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_23_
		//============================================== BEGIN: Fill Orthogonal Basis m.============================================
			get_complex_divided <<< 3, 1 >>> (	(const float *)dev_actual_residual + maxiter,
								(hipComplex *)dev_H_ + (GMRES_i + 1) * maxiter + GMRES_i,
								(float *)dev_actual_residual + maxiter);
			cudacheckSYN();
			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_24_

			cublascall(hipblasCsscal(		(hipblasHandle_t) *handle_p, N * N,
								(const float           *)dev_actual_residual + maxiter,
								(hipComplex       *)dev_orthogonal_basis + (GMRES_i + 1) * N * N, 1));
			cudacheckSYN();
			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_25_
		//===============================    END: Fill Orthogonal Basis m.  ===========================================
		//============================================== Begin: Least Squares Step =========================================================
		//================================ BEGIN: MATMUL (H_temp=Jtotal * H) ==============================================
			cublascall(cublasCgemm3m(	(hipblasHandle_t)*handle_p,
							HIPBLAS_OP_N,
							HIPBLAS_OP_N,
							(unsigned int)GMRES_i + 1,
							(unsigned int)GMRES_i + 2,
							(unsigned int)GMRES_i + 2,
							(const hipComplex *)dev_alpha,
							(hipComplex *)dev_H_, (unsigned int)maxiter,
							(hipComplex *)dev_Jtotal,      (unsigned int)maxiter_plus_1,
							(const hipComplex *)dev_beta,
							(hipComplex *)dev_Htemp, (unsigned int)GMRES_i + 1));
			cudacheckSYN();
			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_26_
		//================================== END: MATMUL (H_temp=Jtotal * H) ===============================================
		//================================================ END: Least Squares Step =========================================================
		//============================================= BEGIN: Create Givens_Rotation_Matrix ========================================
			set_4_Givens_rotation_matrix_elements_kernel <<< 8, 1 >>> (	(hipComplex *)dev_Htemp,
											maxiter_plus_1,
											(hipComplex *)dev_Givens_rotation_0,
											(hipComplex *)dev_Givens_rotation_1,
											(hipComplex *)dev_Givens_rotation_2,
											(hipComplex *)dev_Givens_rotation_3,
											GMRES_i + 1);
			cudacheckSYN();
			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_27_
		//=============================================== END: Create Givens_Rotation_Matrix ========================================
		//================================================== BEGIN: Jtotal = J*Jtotal =================================================
			cublascall(cublasCgemm3m(	(hipblasHandle_t)*handle_p,
							HIPBLAS_OP_N,
							HIPBLAS_OP_N,
							(unsigned int)GMRES_i + 2,
							(unsigned int)GMRES_i + 2,
							(unsigned int)GMRES_i + 2,
							(const hipComplex *)dev_alpha,
							(hipComplex *)dev_Jtotal,          (unsigned int)maxiter_plus_1,
							(hipComplex *)dev_Givens_rotation + (maxiter - 1 - GMRES_i) * (maxiter_plus_1 + 1), (unsigned int)maxiter_plus_1,
							(const hipComplex *)dev_beta,
							(hipComplex *)dev_Jtotal,          (unsigned int)maxiter_plus_1));
			cudacheckSYN();
			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_28_
		//==================================================== END: Jtotal = J*Jtotal =================================================
		//===================================================== BEGIN: Update residual ======================================================
			cudacall(hipMemcpy(&h_Jtotal_0j, dev_Jtotal + maxiter_plus_1 * (GMRES_i + 1), sizeof(hipComplex), hipMemcpyDeviceToHost));
			h_actual_residual = h_residual_initial * sqrt( (pow((float)h_Jtotal_0j.x, 2.0f) + pow((float)h_Jtotal_0j.y, 2.f)));
			cudacall(hipMemcpyAsync(dev_actual_residual + GMRES_i + 1, &h_actual_residual, sizeof(float), hipMemcpyHostToDevice));

			clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_29_
		//======================================================= END: Update residual ======================================================
		}
	//================================================== BEGIN: HH = (Jtotal*H)_resized ==========================================================
		cublascall(cublasCgemm3m(	(hipblasHandle_t)*handle_p,
						HIPBLAS_OP_T,
						HIPBLAS_OP_T,
						(unsigned int)GMRES_i,
						(unsigned int)GMRES_i,
						(unsigned int)GMRES_i + 1,
						(const hipComplex *)dev_alpha,
						(hipComplex *)dev_Jtotal, (unsigned int)maxiter_plus_1,
						(hipComplex *)dev_H_, 	 (unsigned int)maxiter,
						(const hipComplex *)dev_beta,
						(hipComplex *)dev_HH, 	 (unsigned int)GMRES_i));
		cudacheckSYN();
		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_30_
	//===================================================== END: HH = (Jtotal*H)_resized ==========================================================
	//================================================= BEGIN: cc = Jtotal * norm_res_vec =========================================================
		set_cc_kernel <<< GMRES_i, 1 >>> (	(hipComplex *)dev_cc,
							(hipComplex *)dev_Jtotal,
							(float *)dev_actual_residual,
							maxiter_plus_1);
		cudacheckSYN();
		clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_31_
	//=================================================== END: cc = Jtotal * norm_res_vec =========================================================
		if (GMRES_i > 0)
		{
			if (GMRES_i < 2)
			{
				get_new_solution_kernel <<< 1, 1 >>> (	(hipComplex *)dev_cc,
									(hipComplex *)dev_HH);	
				cudacheckSYN();

				get_solution_kernel <<< gridsize, blocksize >>> (	(hipComplex *)dev_solution,
											(hipComplex *)dev_cc,
											(hipComplex *)dev_orthogonal_basis);
				cudacheckSYN();
			}
			else
			{
			//============================================ BEGIN: Find solution to the LES(cc_new) for HH*cc_new=cc ============================================
				cuSolve_LES(	(hipComplex *)dev_HH,
						(hipComplex *)dev_cc,
						GMRES_i,
						*cusolverH_p,
						(hipComplex *)dev_buffer_LES_cc,
						(int *)dev_Ipiv,
						(int *)dev_info,
						(timespec *)h_computation_times,
						(unsigned int *)&clock_i);
			//============================================ END: Find solution to the LES(cc_new) for HH*cc_new=cc ===========================================
			//============================================ BEGIN: x = x0 + V * cc ===========================================
				for(unsigned int j = 0; j < GMRES_i; j++)
				{
					add_kernel <<< gridsize, blocksize >>> ((hipComplex *)dev_solution,
										(hipComplex *)dev_orthogonal_basis + j * N * N,
										(hipComplex *)dev_cc + j);
				//	cudacheckSYN();
					clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_34_
				}
				cudacheckSYN();
			}
		}
	}
	*GMRES_n	 = GMRES_i;
	*h_res_vs_tol_p	 = (h_actual_residual > tolerance);

	clock_gettime(CLOCK_REALTIME, h_computation_times + clock_i++); //_35_
}


void cuSolve_LES(	hipComplex *dev_A, 
			hipComplex *dev_B, 
			const int m, 
			hipsolverHandle_t cusolverH, 
			hipComplex *dev_work, 
			int *dev_Ipiv, 
			int *dev_info, 
			timespec *h_computation_times, 
			unsigned int *clock_i_p)
/*	dev_work <- malloc(SIZE_OF_WORKSPACE * sizeof(hipComplex)) 	device workspace for getrf */
/*	lwork size of workspace  <- #define SIZE_OF_WORKSPACE (N >= 1024)?N/1024*100:100 */
/*	dev_Ipiv <- malloc(maxiter * sizeof(int)) 			pivoting sequence */
/* 	dev_info <- malloc(sizeof(int))					error info */
{
	const int lda = m;
	const int ldb = m;
	int h_info = 0;


	clock_gettime(CLOCK_REALTIME, h_computation_times + (*clock_i_p)++); //_32_

	/* step 2: LU factorization */
	cusolvercall(hipsolverDnCgetrf(      cusolverH,
					    m,
					    m,
					    dev_A,
					    lda,
					    dev_work,
					    dev_Ipiv,
					    dev_info));
	cudacheckSYN();

	cudacall(hipMemcpy(&h_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));

	if ( h_info != 0 )
	{
		printf("hipsolverDnCgetrf failed!\tinfo = %i\n", h_info);
		exit(1);
	}

	cusolvercall(hipsolverDnCgetrs(  cusolverH,
					HIPBLAS_OP_N,
					m,
					1, /* nrhs */
					dev_A,
					lda,
					dev_Ipiv,
					dev_B,
					ldb,
					dev_info));
	cudacheckSYN();

	cudacall(hipMemcpy(&h_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));

	if ( h_info != 0 )
	{
		printf("hipsolverDnCgetrs failed!\tinfo = %i\n", h_info);
		exit(1);
	}

	clock_gettime(CLOCK_REALTIME, h_computation_times + (*clock_i_p)++); //_33_
}

void show_norm(	const char *description, 
		hipblasHandle_t handle, 
		const hipComplex *dev_array, 
		const unsigned int size_array)
{
	float h_residual_0;
	hipblasPointerMode_t mode_current;

	cublascall(hipblasGetPointerMode((hipblasHandle_t)handle, (hipblasPointerMode_t *)&mode_current));
	cublascall(hipblasSetPointerMode((hipblasHandle_t)handle, HIPBLAS_POINTER_MODE_HOST));
	cublascall(hipblasScnrm2((hipblasHandle_t)handle,
					size_array,
		                        (const hipComplex *)dev_array, 1,
					(float  *)&h_residual_0));

	fprintf(stderr, "norm(%s) = %6.12f\n", description, h_residual_0);

	cublascall(hipblasSetPointerMode((hipblasHandle_t)handle, mode_current));
}


void show_norm_F(	const char *description, 
			hipblasHandle_t handle, 
			const float *dev_array, 
			const unsigned int size_array)
{
	float h_residual_0;
	hipblasPointerMode_t mode_current;

	cublascall(hipblasGetPointerMode((hipblasHandle_t)handle, (hipblasPointerMode_t *)&mode_current));
	cublascall(hipblasSetPointerMode((hipblasHandle_t)handle, HIPBLAS_POINTER_MODE_HOST));
	cublascall(hipblasSnrm2((hipblasHandle_t)handle,
					size_array,
		                        (const float *)dev_array, 1,
					(float  *)&h_residual_0));

	fprintf(stderr, "norm(%s) = %6.12f\n", description, h_residual_0);

	cublascall(hipblasSetPointerMode((hipblasHandle_t)handle, mode_current));
}
