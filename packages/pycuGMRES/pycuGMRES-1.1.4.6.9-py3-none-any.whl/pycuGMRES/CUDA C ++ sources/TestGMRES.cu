#include "hip/hip_runtime.h"
//#include <sstream>
//#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <string>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "GMRES_kernels.cuh"
#include "saveGPU.cuh"
#include "Fast_matvec.cuh"
#include "pycuFunctions.cuh"
#include "GMRES.cuh"

unsigned int *get_n_timestamps_array_improved(unsigned int max_maxiter);

#include "GMRES_old_kernels.cuh"
#include "GMRES_old.cuh"


void pycuTestGMRES()
{		
	char buffer[1024];
	float tolerance = 0.001f;//0.2f;

	unsigned int rep_st = 0;
	unsigned int rep_en = 0;

	unsigned int pow_st = 10;
	unsigned int pow_en = 10;

	unsigned int min_maxiter = 30;
	unsigned int max_maxiter = 30;

	unsigned int *n_timestamps_array = get_n_timestamps_array_improved((unsigned int)max_maxiter + 1);


	devSubsidiary dev_subs[1];


	hipComplex **p_h_anal_sols = (hipComplex **) malloc((pow_en - pow_st + 1) * sizeof(hipComplex *));
	bool **p_h_masks = (bool **) malloc((pow_en - pow_st + 1) * sizeof(bool *));
	hipComplex **p_h_gamma_arrays = (hipComplex **) malloc((pow_en - pow_st + 1) * sizeof(hipComplex *));

	for (unsigned int pow_cur = pow_st; pow_cur < pow_en + 1; pow_cur ++)
	{
		unsigned int N = 1 << pow_cur;
		p_h_anal_sols[pow_cur - pow_st] = (hipComplex *) malloc( N * N * sizeof(hipComplex) );

		std::string line;
		sprintf(buffer, "/media/linux/4db3d51d-3503-451d-aff7-07e3ce95927e/Archive/Input/analytical_solution_%u.txt", N);
		std::ifstream analytical_solution_file (buffer);//Python_analytical_solution_%u
		if (analytical_solution_file.is_open())
		{
			unsigned int index = 0;
			while ( getline (analytical_solution_file, line) )
			{
				std::istringstream in_string_stream(line);

				in_string_stream >> p_h_anal_sols[pow_cur - pow_st][index].x >> p_h_anal_sols[pow_cur - pow_st][index].y;

				index++;
	
			}
			analytical_solution_file.close();
		}
		else
		{
			fprintf(stderr, "Unable to open file: %s\n", buffer);
			exit(1);
		}


		p_h_masks[pow_cur - pow_st] = (bool *) malloc(N * N * sizeof(bool));

		sprintf(buffer, "/media/linux/4db3d51d-3503-451d-aff7-07e3ce95927e/Archive/Input/cylinder_%u.txt", N);
		std::ifstream myfile (buffer);
		if (myfile.is_open())
		{
			unsigned int index = 0;
			while ( getline (myfile,line) )
			{
				p_h_masks[pow_cur - pow_st][index++] = (line == "1");
			}
			myfile.close();
		}
		else {
			fprintf(stderr, "Unable to open file: %s\n", buffer);
			exit(1);
		}


		p_h_gamma_arrays[pow_cur - pow_st] = (hipComplex *)malloc((2 * N - 1) * (2 * N - 1) * sizeof(hipComplex));
	
		sprintf(buffer, "/media/linux/4db3d51d-3503-451d-aff7-07e3ce95927e/Archive/Input/G_prep_%u.txt", N);
		get_array_C_to_CPU((hipComplex *)p_h_gamma_arrays[pow_cur - pow_st], (const char *)buffer);
	}


	for (unsigned int repetition_i = rep_st; repetition_i < rep_en + 1; repetition_i ++)
	{	// int maxiter = 28;
		for (unsigned int maxiter = min_maxiter; maxiter < max_maxiter + 1; maxiter ++)
		{
			for (unsigned int pow_cur = pow_st; pow_cur < pow_en + 1; pow_cur = pow_cur + 5) //Characteristic size of square matrix
			{
				unsigned int N = 1 << pow_cur;

				fprintf(stderr, "%i\n", N);
	
				dim3 blocks(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
				dim3 threads(Q, Q);
				hipfftHandle plan;
				hipblasHandle_t handle;
				cublascall(hipblasCreate(&handle));
				cufftcall(hipfftPlan2d(&plan, 2 * N - 1, 2 * N - 1, HIPFFT_C2C));
				hipStream_t stream = NULL;
				hipsolverHandle_t cusolverH = NULL;
				cusolvercall(hipsolverDnCreate(&cusolverH));
				cudacall(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
				cusolvercall(hipsolverSetStream(cusolverH, stream));

				bool *dev_mask;
				bool *h_mask = p_h_masks[pow_cur - pow_st];
				bool h_res_vs_tol = true;
				hipComplex *h_gamma_array = p_h_gamma_arrays[pow_cur - pow_st];
				hipComplex *h_analytical_solution = p_h_anal_sols[pow_cur - pow_st];
				hipComplex *dev_gamma_array;
				hipComplex *dev_analytical_solution;
				hipComplex *dev_solution;
				float *dev_actual_residual;
				float h_result = 0.f;
				float h_norm_analytical_solution = 0.f;
				unsigned int GMRES_n = 0;
				timespec *h_computation_times = (timespec *) malloc(n_timestamps_array[maxiter] * sizeof(timespec));
				cudacall(hipSetDevice(0));

				cudacall(hipMalloc((void**)&dev_mask, N * N * sizeof(bool)));
				cudacall(hipMalloc((void**)&dev_solution, N * N * sizeof(hipComplex)));
				cudacall(hipMalloc((void**)&dev_analytical_solution, N * N * sizeof(hipComplex)));




				cudacall(hipMemcpy(dev_analytical_solution, h_analytical_solution, N * N * sizeof(hipComplex), hipMemcpyHostToDevice));


				cublascall(hipblasScnrm2(handle, N * N,
							(const hipComplex *)dev_analytical_solution, 1, 
							(float  *)&h_norm_analytical_solution));


				cudacall(hipMemcpy(dev_mask, h_mask, N * N * sizeof(bool), hipMemcpyHostToDevice));

			//	get_gamma_array((hipComplex **)&dev_gamma_array, (hipfftHandle)plan);
			//	cudacall(hipMemcpy(h_gamma_array, dev_gamma_array, (2 * N - 1) * (2 * N - 1) * sizeof(hipComplex), hipMemcpyDeviceToHost));
			//==================================== Begin: get_gamma_array connected to MKL 2D Green's function values in Bessel function =========================
				cudacall(hipMalloc((void**)&dev_gamma_array,  (2 * N - 1) * (2 * N - 1) * sizeof(hipComplex)));
				cudacall(hipMemcpy(dev_gamma_array, h_gamma_array, (2 * N - 1) * (2 * N - 1) * sizeof(hipComplex), hipMemcpyHostToDevice));

				cufftcall(hipfftExecC2C(plan, (hipComplex *)dev_gamma_array, (hipComplex *)dev_gamma_array, HIPFFT_FORWARD));
				cudacheckSYN();
			//==================================== End: get_gamma_array connected to MKL 2D Green's function values in Bessel function =========================

				time_t clock_time;
				float diff_time = 0.f;
				float diff_average = 0.f;
				hipComplex alpha;
				alpha.x = -1.f;
				alpha.y = 0.f;
				const hipComplex *p_alpha = &alpha;

				{

					cudacall(hipMalloc((void**)&dev_actual_residual, (maxiter + 1) * sizeof(float)));

					const hipError_t *allocation_result = pycuGetSubsidiary(
								(devSubsidiary *)dev_subs, 
								N, 
								maxiter);

					fprintf(stderr, "Allocation memory (vs. 0): %d\n", allocation_result[0]);

					hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
					fprintf(stderr, "maxiter = %i\trepetition_i = %i\n", maxiter, repetition_i);


					const float wavenumber = 2*3.14f/(N/6.f);
					const float eps_in = 2.25f;
					const float eps_ex = 1.00f;
					const float wavenumber_ref = wavenumber * sqrt(eps_ex);


					init_x0_kernel <<< blocks, threads >>> ((hipComplex *)dev_solution, -1.f, N, wavenumber_ref);
					cudacheckSYN();

					memset(h_computation_times, 0, n_timestamps_array[maxiter] * sizeof(timespec));

					clock_time = clock();
/*
					Fast_GMRES_with_CUDA(	(const hipComplex *)dev_gamma_array,
								(const bool *)dev_mask,
								(hipComplex *)dev_solution,
								(float **)&dev_actual_residual,
								(unsigned int *)&GMRES_n,
								(hipfftHandle)plan,
								(hipblasHandle_t *)&handle,
								tolerance, false, 0,
								(bool *)&h_res_vs_tol,
 								maxiter,
								(hipsolverHandle_t)cusolverH,
								(timespec *)h_computation_times, N);

*/					pycuGMRESold(	
							    (bool *)dev_mask,
							    (hipComplex *)dev_solution,
							    false,
							    0,
									-1.f,
							    maxiter,
							    tolerance,
							    (unsigned int *)&GMRES_n,
							    (float *)dev_actual_residual,
							    (bool *)&h_res_vs_tol,
							    N,
							    (hipComplex *)dev_gamma_array,
							    plan,
							    (hipblasHandle_t *)&handle,
							    (hipsolverHandle_t *)&cusolverH,
							    (devSubsidiary *)dev_subs,
							    (timespec *)h_computation_times,
							    wavenumber,
									eps_in,
									eps_ex
						);

					diff_time = (float)(clock() - clock_time) / (float)(CLOCKS_PER_SEC);

					pycuDestroySubsidiary((devSubsidiary *)dev_subs);
				}

				{
					fprintf(stderr, "Files writing\n");
		
					sprintf(buffer, "time_%u/solution_sample", N);
					save_test_GPU((char *)buffer, (hipComplex *)dev_solution, maxiter * 100 + repetition_i, N * N);
					fprintf(stderr, "diff_time = %f\n", diff_time);

					sprintf(buffer, "time_%u/maxiter", N);
					save_test_F_CPU((char *)buffer, (float *)&diff_time, maxiter * 100 + repetition_i, 1);
					sprintf(buffer, "time_%u/residual", N);
					save_test_F_GPU((char *)buffer, (float *)dev_actual_residual + GMRES_n, maxiter * 100 + repetition_i, 1);
					sprintf(buffer, "time_%u/times", N);
					save_test_timespec_CPU((char *)buffer, (timespec *)h_computation_times, maxiter * 100 + repetition_i, n_timestamps_array[maxiter]);

					hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

					cublascall(hipblasScnrm2(handle, N * N,
								    (const hipComplex *)dev_solution, 1, (float  *)&h_result));

					fprintf(stderr, "Norm of solution = %f\n", h_result);


					cublascall(hipblasCaxpy(handle, N * N,
								   (const hipComplex *)p_alpha,
								   (const hipComplex *)dev_analytical_solution, 1,
								   (hipComplex *)dev_solution, 1));


					cublascall(hipblasScnrm2(handle, N * N,
								    (const hipComplex *)dev_solution, 1, (float  *)&h_result));

					fprintf(stderr, "Norm of diff = %f\n", h_result);

					h_result = h_result / h_norm_analytical_solution;

					fprintf(stderr, "File relative_error writing\t%f\n", h_result);
					sprintf(buffer, "time_%u/relative_error", N);
					save_test_F_CPU((char *)buffer, (float *)&h_result, maxiter * 100 + repetition_i, 1);
				}

				fprintf(stderr, "diff = %f\n", diff_average);

	//			saveGPUrealtxt_C(dev_solution, "/output/solution.txt", N * N);

				cudacall(hipFree((bool *)dev_mask));
				cudacall(hipFree((hipComplex *)dev_solution));
				cudacall(hipFree((hipComplex *)dev_gamma_array));
				cudacall(hipFree((hipComplex *)dev_analytical_solution));
				cufftcall(hipfftDestroy(plan));
				hipsolverDnDestroy(cusolverH);
				free((timespec *)h_computation_times);
				cublascall(hipblasDestroy(handle));
                                cudacall(hipFree((float *)dev_actual_residual));
			}
		}
	}

	free(n_timestamps_array);
}


unsigned int get_n_timestamps_val_improved(unsigned int maxiter) //Comparables/new
{
    unsigned int n_timestamps  = 1; //short_indexed_text_array = []
    n_timestamps ++; //short_indexed_text_array.append("Initialization (malloc)") #_1_ !_
    n_timestamps ++; //short_indexed_text_array.append("G_x_fft_matvec for A*x0") #_2_ !_
    n_timestamps ++; //short_indexed_text_array.append("2D_to_1D for A*x0-x0") #_3_
    n_timestamps ++; //short_indexed_text_array.append("Norm(residual_vec)") #_4_
    n_timestamps ++; //short_indexed_text_array.append("Condition to iterate") #_5_ !_
    n_timestamps ++; //short_indexed_text_array.append("Residual_normalization & set_a,b") #_6_
    
    unsigned int GMRES_i = 1;
    
    if (1)
    {
        n_timestamps ++; //short_indexed_text_array.append("Memset(H, 0)") #_7_ !_
        n_timestamps ++; //short_indexed_text_array.append("G_x_fft_matvec for w=A*v iteration(" + str(GMRES_i) + ")") #_8_
        n_timestamps ++; //short_indexed_text_array.append("2D_to_1D for w=A*v iteration(" + str(GMRES_i) + ")") #_9_
        n_timestamps ++; //short_indexed_text_array.append("H_jk = (V_j, w) iteration(" + str(GMRES_i) + ")") #_10_
        n_timestamps ++; //short_indexed_text_array.append("w = w - H*v iteration(" + str(GMRES_i) + ")") #_11_ !_    
        n_timestamps ++; //short_indexed_text_array.append("H_jj+1 = norm(w) iteration(" + str(GMRES_i) + ")") #_12_    
        n_timestamps ++; //short_indexed_text_array.append("1/H_jj+1 iteration(" + str(GMRES_i) + ")") #_13_    
        n_timestamps ++; //short_indexed_text_array.append("w = w/H_jj+1 iteration(" + str(GMRES_i) + ")") #_14_
        n_timestamps ++; //short_indexed_text_array.append("Set(J) iteration(" + str(GMRES_i) + ")") #_15_ !_
        n_timestamps ++; //short_indexed_text_array.append("Set(Jtotal) iteration(" + str(GMRES_i) + ")") #_16_ !_
        n_timestamps ++; //short_indexed_text_array.append("Update residual iteration(" + str(GMRES_i) + ")") #_17_ !_
        
        for (GMRES_i = 1; GMRES_i < maxiter; GMRES_i ++)
        {  
            n_timestamps ++; //short_indexed_text_array.append("Condition_check iteration(" + str(GMRES_i) + ")") #_18_
            n_timestamps ++; //short_indexed_text_array.append("G_x_fft_matvec for w=A*v iteration(" + str(GMRES_i) + ")") #_19_        
            n_timestamps ++; //short_indexed_text_array.append("2D_to_1D for w=A*v iteration(" + str(GMRES_i) + ")") #_20_     
                
            for (unsigned int j = 0; j < GMRES_i + 1; j ++)
            {
                n_timestamps ++; //short_indexed_text_array.append("H_jk = (V_j, w) iteration(" + str(GMRES_i) + ", j = " + str(j) + ")") #_21_
                n_timestamps ++; //short_indexed_text_array.append("w = w - H_jk * V_j iteration(" + str(GMRES_i) + ", j = " + str(j) + ")") #_22_  
            }       
                
            n_timestamps ++; //short_indexed_text_array.append("H_jj+1 = norm(w) iteration(" + str(GMRES_i) + ")") #_23_
            n_timestamps ++; //short_indexed_text_array.append("1/H_jj+1 iteration(" + str(GMRES_i) + ")") #_24_
            n_timestamps ++; //short_indexed_text_array.append("w = w/H_jj+1 iteration(" + str(GMRES_i) + ")") #_25_    
            n_timestamps ++; //short_indexed_text_array.append("H_temp=Jtotal * H iteration(" + str(GMRES_i) + ")") #_26_
            n_timestamps ++; //short_indexed_text_array.append("Set(J) iteration(" + str(GMRES_i) + ")") #_27_ !_
            n_timestamps ++; //short_indexed_text_array.append("Jtotal = J*Jtotal iteration(" + str(GMRES_i) + ")") #_28_
            n_timestamps ++; //short_indexed_text_array.append("Update residual iteration(" + str(GMRES_i) + ")") #_29_ !_
        }
    }
            
    n_timestamps ++; //short_indexed_text_array.append("HH = Jtotal * H") #_30_
    n_timestamps ++; //short_indexed_text_array.append("cc <- Jtotal") #_31_
    n_timestamps ++; //short_indexed_text_array.append("Initialize_small_LES(HH, cc)") #_32_
    n_timestamps ++; //short_indexed_text_array.append("Process_small_LES(HH, cc)") #_33_
    
    for (unsigned int j = 0; j < GMRES_i; j++)
    {        
        n_timestamps ++; //short_indexed_text_array.append("Add iteration(j = " + str(j) + ")") #_34_
    }
        
    n_timestamps ++; //short_indexed_text_array.append("set(Output_p)") #_35_        
    n_timestamps ++; //short_indexed_text_array.append("Free in postprocessing") #_36_
    
    return n_timestamps;
}


unsigned int *get_n_timestamps_array_improved(unsigned int max_maxiter)
{
    unsigned int *n_timestamps_array = (unsigned int *)malloc(max_maxiter * sizeof(unsigned int));

    for (unsigned int maxiter = 0; maxiter < max_maxiter; maxiter ++)
    {
        n_timestamps_array[maxiter] = get_n_timestamps_val_improved((unsigned int)maxiter);
    }
    return n_timestamps_array;
}
